#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C" {

#include <stdio.h>

__host__ void h_relu(float* a, float* b, int n) {
    for (int i = 0; i < n; i++) {
        if (a[i] > 0) {
            b[i] = a[i];
        }
        else {
            b[i] = 0;
        }
    }
}


__global__ void d_relu(float *a, float *b, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        if (a[i] > 0) {
            b[i] = a[i];
        }
        else {
            b[i] = 0;
        }
    }
}


__global__ void d_fill_uniform(
    float *a, int n, float r, unsigned long long seed) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        hiprandState_t state;
        hiprand_init(seed, i, 0, &state);

        a[i] = -r + 2 * r * hiprand_uniform(&state);
    }
}


float compare(float *a, float *b, int n, float eps) {
    float diff = 0;

    for (int i = 0; i < n; i++) {
        diff = fabs(a[i] - b[i]);
        if (diff >= eps) {
            return diff;
        }
    }

    return diff;
}

}

#ifndef REDEFINE
    #define VEC_LEN 51200000
    #define VEC_LEN_INC 512000
    #define CHECK_FIRST 51200
    #define BLOCK_SIZE 128
    #define FNAME_STAMPS "timings.stmp"
    #define PRECISION 10e-10
    #define SEED 27
    #define VEC_MAX_ABS_VAL 101
#endif

#define VEC_MEM_SIZE (VEC_LEN * sizeof(float))
#define ts_to_ms(ts) (ts.tv_sec * 10e3 + ts.tv_nsec * 10e-6)
#define calc_grid_size(m) ((m + BLOCK_SIZE - 1) / BLOCK_SIZE)


int main() {
    float *h_a __attribute__ ((aligned (64)));
    float *h_b __attribute__ ((aligned (64)));
    float *h_c __attribute__ ((aligned (64)));

    h_a = (float*)malloc(VEC_MEM_SIZE);
    h_b = (float*)malloc(VEC_MEM_SIZE);
    h_c = (float*)malloc(VEC_MEM_SIZE);

    float *d_a, *d_b;
    hipMalloc((void**)&d_a, VEC_MEM_SIZE);
    hipMalloc((void**)&d_b, VEC_MEM_SIZE);

    d_fill_uniform<<<calc_grid_size(VEC_LEN), BLOCK_SIZE>>>(
        d_a, VEC_LEN, VEC_MAX_ABS_VAL, SEED);
    hipMemcpy(h_a, d_a, VEC_MEM_SIZE, hipMemcpyDeviceToHost);

    h_relu(h_a, h_b, CHECK_FIRST);
    d_relu<<<calc_grid_size(CHECK_FIRST), BLOCK_SIZE>>>(d_a, d_b, CHECK_FIRST);
    hipMemcpy(h_c, d_b, CHECK_FIRST * sizeof(float), hipMemcpyDeviceToHost);

    if (compare(h_b, h_c, CHECK_FIRST, PRECISION) > PRECISION) {
        printf("Panic!\n");
        return -1;
    }

    float h_time;
    timespec h_start, h_stop;

    float d_time;
    hipEvent_t d_start, d_stop;
    hipEventCreate(&d_start);
    hipEventCreate(&d_stop);

    FILE* file = fopen(FNAME_STAMPS, "w");
    fprintf(file, "Vector Length, CPU Time, GPU Time\n");

    for (int m = VEC_LEN_INC; m <= VEC_LEN; m += VEC_LEN_INC) {
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &h_start);
        h_relu(h_a, h_b, m);
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &h_stop);
        h_time = (ts_to_ms(h_stop) - ts_to_ms(h_start)); // time in ms

        hipEventRecord(d_start);
        d_relu<<<calc_grid_size(m), BLOCK_SIZE>>>(d_a, d_b, m);
        hipEventRecord(d_stop);
        hipEventSynchronize(d_stop);
        hipEventElapsedTime(&d_time, d_start, d_stop); // time in ms

        fprintf(file, "%d, %f, %f\n", m, h_time, d_time);
    }

    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);

    fclose(file);

    return 0;
}
